#include "hip/hip_runtime.h"
#include <cistem_config.h>

#include "../../gpu/gpu_core_headers.h"
#include "../../gpu/GpuImage.h"

// #define ELECTRON_DOSE_DEBUG_PRINT

class StealStdoutBackFromWX {

    //   private:

  public:
    StealStdoutBackFromWX(const char* file_name) : _my_file(fopen(file_name, "w")) {
        _store_original_stdout_ptr = stdout;
        stdout                     = _my_file;
    };

    ~StealStdoutBackFromWX( ) { RestoreStdoutToWX( ); };

    void FlushCapturedStdout( ) { fflush(stdout); };

    inline void RestoreStdoutToWX( ) { stdout = _store_original_stdout_ptr; };

    FILE* _store_original_stdout_ptr;
    FILE* _my_file;
};

__device__ __inline__ float
ReturnCriticalDose(float spatial_frequency, float voltage_scaling_factor) {

    return (cistem::electron_dose::critical_dose_a * powf(spatial_frequency, cistem::electron_dose::reduced_critical_dose_b) + cistem::electron_dose::critical_dose_c) * voltage_scaling_factor;
};

__device__ __inline__ float
ReturnDoseFilter(float dose_at_end_of_frame, float critical_dose) {
    return expf((-0.5 * dose_at_end_of_frame) / critical_dose);
};

__device__ __inline__ float
ReturnCummulativeDoseFilter(float dose_at_start_of_exposure, float dose_at_end_of_exosure, float critical_dose) {
    // The integrated exposure. Included in particular for the matched filter.
    // Calculated on Wolfram Alpha = integrate exp[ -0.5 * (x/a) ] from x=0 to x=t
    return 2.0f * critical_dose * (exp((-0.5 * dose_at_start_of_exposure) / critical_dose) - exp((-0.5 * dose_at_end_of_exosure) / critical_dose)) / dose_at_end_of_exosure;
};

__global__ void
ApplyDoseFilterKernel(float2**    image_data,
                      float       pre_exposure,
                      const float dose_per_frame,
                      float2*     output_data,
                      const float voltage_scaling_factor,
                      const float fourier_voxel_size_x,
                      const float fourier_voxel_size_y,
                      const int   pixel_pitch,
                      const int   NY,
                      const int   NZ,
                      const int   physical_index_of_first_negative_frequency_y) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if ( x >= pixel_pitch )
        return;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if ( y >= NY )
        return;

    const int address = x + y * pixel_pitch;
    // logical fourier index
    y        = (y >= physical_index_of_first_negative_frequency_y) ? (y - NY) : y;
    float ky = float(y) * fourier_voxel_size_y;
    ky *= ky;

    float kx = float(x) * fourier_voxel_size_x;
    kx       = kx * kx + ky;

    float real_sum = 0.f;
    float imag_sum = 0.f;
    float filter_coeff;

    for ( int z = 0; z < NZ; z++ ) {
#ifdef ELECTRON_DOSE_DEBUG_PRINT
        printf("Index is %i, %i, %i, %i \n", x, y, z, address); //, image_data[z].complex_values_gpu[address].x);
#endif

        filter_coeff = ReturnDoseFilter(pre_exposure, ReturnCriticalDose(kx, voltage_scaling_factor));
        real_sum += image_data[z][address].x * filter_coeff;
        imag_sum += image_data[z][address].y * filter_coeff;
        pre_exposure += dose_per_frame;
    }

    // #ifdef ELECTRON_DOSE_DEBUG_PRINT
    //     printf("Index is %i, %i, %i, %i \n", x, y, 0, address); //, image_data[z].complex_values_gpu[address].x);
    // #endif

    if ( address == 0 ) {
        output_data[0].x = 1.0f;
        output_data[0].y = 0.0f;
    }
    else {
        output_data[address].x = real_sum;
        output_data[address].y = imag_sum;
    }
};

__global__ void ApplyDoseFilterAndRestorePowerKernel(float2**    image_data,
                                                     float       pre_exposure,
                                                     const float dose_per_frame,
                                                     float2*     output_data,
                                                     const float voltage_scaling_factor,
                                                     const float fourier_voxel_size_x,
                                                     const float fourier_voxel_size_y,
                                                     const int   pixel_pitch,
                                                     const int   NY,
                                                     const int   NZ,
                                                     const int   physical_index_of_first_negative_frequency_y) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if ( x >= pixel_pitch )
        return;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if ( y >= NY )
        return;

    const int address = x + y * pixel_pitch;
    // logical fourier index
    y        = (y >= physical_index_of_first_negative_frequency_y) ? (y - NY) : y;
    float ky = float(y) * fourier_voxel_size_y;
    ky *= ky;

    float kx = float(x) * fourier_voxel_size_x;
    kx       = kx * kx + ky;

    float real_sum = 0.f;
    float imag_sum = 0.f;
    float filter_coeff;
    float sum_of_squares = 0.f;
    for ( int z = 0; z < NZ; z++ ) {
        filter_coeff = ReturnDoseFilter(pre_exposure, ReturnCriticalDose(kx, voltage_scaling_factor));
        sum_of_squares += (filter_coeff * filter_coeff);
        real_sum += image_data[z][address].x * filter_coeff;
        imag_sum += image_data[z][address].y * filter_coeff;
        pre_exposure += dose_per_frame;
    }

    // This should never be zero
    sum_of_squares = sqrtf(sum_of_squares);

    if ( address == 0 ) {
        output_data[0].x = 1.0f;
        output_data[0].y = 0.0f;
    }
    else {
        output_data[address].x = real_sum / sum_of_squares;
        output_data[address].y = imag_sum / sum_of_squares;
    }
};

template <>
void ElectronDose::CalculateDoseFilterAs1DArray(std::vector<GpuImage>& ref_image, float2* output_data, float pre_exposure, float exposure_per_frame, bool restore_power) {

    // Different than the CPU implementation, dose_start is assumed to be pre_exposure and dose per frame = dose_finish
    ref_image[0].ReturnLaunchParameters(ref_image[0].dims, false);

#ifdef ELECTRON_DOSE_DEBUG_PRINT
    StealStdoutBackFromWX my_stdout("/tmp/gpulog_class.txt");
#endif
    ref_image[0].ptr_array_32fc.resize(ref_image.size( ));
    for ( int iPtr = 0; iPtr < ref_image.size( ); iPtr++ ) {
        ref_image[0].ptr_array_32fc.SetPointer((float2*)ref_image[iPtr].complex_values_gpu, iPtr);
    }
    if ( restore_power ) {
        precheck
                ApplyDoseFilterAndRestorePowerKernel<<<ref_image[0].gridDims, ref_image[0].threadsPerBlock, 0, hipStreamPerThread>>>(ref_image[0].ptr_array_32fc.ptr_array,
                                                                                                                                      pre_exposure,
                                                                                                                                      exposure_per_frame,
                                                                                                                                      output_data,
                                                                                                                                      voltage_scaling_factor,
                                                                                                                                      ref_image[0].fourier_voxel_size.x / pixel_size,
                                                                                                                                      ref_image[0].fourier_voxel_size.y / pixel_size,
                                                                                                                                      ref_image[0].dims.w / 2,
                                                                                                                                      ref_image[0].dims.y,
                                                                                                                                      ref_image.size( ),
                                                                                                                                      ref_image[0].physical_index_of_first_negative_frequency.y);
        postcheck
    }
    else {
        precheck
                ApplyDoseFilterKernel<<<ref_image[0].gridDims, ref_image[0].threadsPerBlock, 0, hipStreamPerThread>>>(ref_image[0].ptr_array_32fc.ptr_array,
                                                                                                                       pre_exposure,
                                                                                                                       exposure_per_frame,
                                                                                                                       output_data,
                                                                                                                       voltage_scaling_factor,
                                                                                                                       ref_image[0].fourier_voxel_size.x / pixel_size,
                                                                                                                       ref_image[0].fourier_voxel_size.y / pixel_size,
                                                                                                                       ref_image[0].dims.w / 2,
                                                                                                                       ref_image[0].dims.y,
                                                                                                                       ref_image.size( ),
                                                                                                                       ref_image[0].physical_index_of_first_negative_frequency.y);
#ifdef ELECTRON_DOSE_DEBUG_PRINT
        hipDeviceSynchronize( );
        hipDeviceReset( );
        my_stdout.FlushCapturedStdout( );
        exit(0);
#endif

        postcheck
    }

    //	MyDebugAssertTrue(ref_image->logical_z_dimension == 1, "Reference Image is a 3D!");
}