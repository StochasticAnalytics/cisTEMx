#include "hip/hip_runtime.h"
#include "../../../gpu/gpu_core_headers.h"
#include "test_gpu_core_headers.cuh"

__global__ void
test_complex_add_kernel(Complex* a, Complex* b, Complex* output) {
    *output = ComplexAdd(*a, *b);
}

void test_complex_add(Complex* a, Complex* b, Complex* output) {
    test_complex_add_kernel<<<1, 1, 0, hipStreamPerThread>>>(a, b, output);
    cudaErr(hipStreamSynchronize(hipStreamPerThread))
}

__global__ void
test_complex_scale_kernel(Complex* a, float output) {
    ComplexScale(a, output);
}

void test_complex_scale(Complex* a, float scalar) {
    test_complex_scale_kernel<<<1, 1, 0, hipStreamPerThread>>>(a, scalar);
    cudaErr(hipStreamSynchronize(hipStreamPerThread))
}

__global__ void
test_complex_scale_kernel(Complex& a, Complex& output, float scalar) {
    output = ComplexScale(a, scalar);
}

Complex test_complex_scale(Complex& a, float scalar) {
    Complex* output;
    Complex  ret_val;
    cudaErr(hipMalloc(&output, sizeof(Complex)));

    test_complex_scale_kernel<<<1, 1, 0, hipStreamPerThread>>>(a, *output, scalar);
    cudaErr(hipMemcpyAsync(&ret_val, output, sizeof(Complex), hipMemcpyDeviceToHost, hipStreamPerThread));
    cudaErr(hipStreamSynchronize(hipStreamPerThread));
    return ret_val;
}